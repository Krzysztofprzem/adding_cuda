#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <windows.h>

using namespace std;

template <typename  T>
hipError_t LeibnitzFormula(T* numbers, int N);

template <typename  T>
hipError_t Sum(T*  numbers, int N, T SumCPU, LARGE_INTEGER TimeCPU);

template <typename  T>
__global__ void LeibnitzFormulaGPU(T* numbers, int N, int offset) {		// <- Nooo mo�e prawie. Chc� wyznaczy� Pi wi�c mno�� jeszcze razy 4
	int index = (blockIdx.x + offset)*blockDim.x + threadIdx.x;
	if (index < N)
		numbers[index] = (index % 2 == 0) ? (T)(4 / (T)(2 * index + 1)) : (T)(-4 / (T)(2 * index + 1));
}

template <typename  T>
__global__ void SumGPU(T* numbers, int N, int offset) {
	int Tindex = threadIdx.x;										// <-wsp�rz�dne dla shared'a
	int Bindex = blockIdx.x + offset;								// <-wsp�rz�dne bloku (miejsca na, kt�re zostanie przeniesiona suma)
	int Gindex = (blockIdx.x + offset) * 2 * blockDim.x + threadIdx.x;	// <-wsp�rz�dne globalne
	extern __shared__ __align__(sizeof(T)) unsigned char s[];
	T *temp = reinterpret_cast<T *>(s);
	if (Gindex + blockDim.x < N)											// <-je�li globalne wsp�rz�dne s� mniejsze od ilosci pr�bek...
		temp[Tindex] = numbers[Gindex] + numbers[Gindex + blockDim.x];		// <-we� warto�� do shared'a
	else if (Gindex < N)
		temp[Tindex] = numbers[Gindex];
	else
		temp[Tindex] = 0;
	__syncthreads();

	for (int i = blockDim.x / 2; i > 0; i /= 2) {	//<- i to po�owa bloku
		if (Tindex < i)
			temp[Tindex] += temp[Tindex + i];
		__syncthreads();
	}
	if (Tindex == 0)					// <-spe�nione tylko dla jednego w�tku w bloku
		numbers[Bindex] = temp[Tindex];	// <-przypisanie warto�ci sumy danego bloku na kom�rk� o indeksie bloku z kt�rego pochodzi
}

int main() {
	// o=====<+>=====<+>=====<+>=====<+>=====<+>=====<+>=====o //
	//					  DODAWANIE FLOAT'�W					   //
	// o=====<+>=====<+>=====<+>=====<+>=====<+>=====<+>=====o //
	int N = 343154432;
	//N /= 2;
	float* numbersf = new float[N];
	hipError_t cudaStatus = LeibnitzFormula(numbersf, N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "LeibnitzFormula failed!");
		return 1;
	}

	cout << "DODAWANIE FLOAT'OW" << endl;
	cout << "N: " << N << endl;


	LARGE_INTEGER StartCPUf, StopCPUf, TimeCPUf;
	LARGE_INTEGER frequency;
	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&StartCPUf);



	float SumCPUf = 0;
	for (int i = 0; i < N; i++) {
		SumCPUf = SumCPUf + numbersf[i];
	}



	QueryPerformanceCounter(&StopCPUf);
	TimeCPUf.QuadPart = StopCPUf.QuadPart - StartCPUf.QuadPart;
	TimeCPUf.QuadPart = TimeCPUf.QuadPart * 1000000 / frequency.QuadPart / 1000;	//ms

	cout << setprecision(50) << "PI=\t\t3.141592653589793238462643383279502884197169399375105820" << endl;
	cout << "SumCPU= \t" << SumCPUf << endl;
	cout << "Czas dodawania na CPU wynosi:" << TimeCPUf.QuadPart << "ms" << endl;
	cout << endl;



	cudaStatus = Sum(numbersf, N, SumCPUf, TimeCPUf);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sum failed!");
		cout << endl << hipGetErrorString(cudaStatus) << endl;
		return 1;
	}


	delete[] numbersf;
	cout << endl;
	cout << endl;
	cout << endl;
	// o=====<+>=====<+>=====<+>=====<+>=====<+>=====<+>=====o //
	//					 DODAWANIE DOUBLE'I					   //
	// o=====<+>=====<+>=====<+>=====<+>=====<+>=====<+>=====o //
	N = N / 2;
	double* numbersd = new double[N];
	cudaStatus = LeibnitzFormula(numbersd, N);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "LeibnitzFormula failed!");
		return 1;
	}
	cout << "DODAWANIE DOUBLE'I" << endl;
	cout << "N: " << N << endl;

	LARGE_INTEGER StartCPUd, StopCPUd, TimeCPUd;
	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&StartCPUd);



	double SumCPUd = 0;
	for (int i = 0; i < N; i++) {
		SumCPUd = SumCPUd + numbersd[i];
	}



	QueryPerformanceCounter(&StopCPUd);
	TimeCPUd.QuadPart = StopCPUd.QuadPart - StartCPUd.QuadPart;
	TimeCPUd.QuadPart = TimeCPUd.QuadPart * 1000000 / frequency.QuadPart / 1000;	//ms

	cout << setprecision(50) << "PI=\t\t3.141592653589793238462643383279502884197169399375105820" << endl;
	cout << "SumCPU= \t" << SumCPUd << endl;
	cout << "Czas dodawania na CPU wynosi: " << TimeCPUd.QuadPart << "ms" << endl;
	cout << endl;




	cudaStatus = Sum(numbersd, N, SumCPUd, TimeCPUd);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Sum failed!");
		cout << endl << hipGetErrorString(cudaStatus) << endl;
		return 1;
	}






	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
template <typename  T>
hipError_t LeibnitzFormula(T* numbers, int N)
{
	T *dev_n = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//for (;;) {						//<-Sprawdzajka do wyznaczania dok�adnej ilo�ci 
	//	cout << "N= " << N << endl;
	cudaStatus = hipMalloc((void**)&dev_n, N * sizeof(T));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//hipFree(dev_n);
	//	N = N + 1;
	//}

	cudaStatus = hipMemcpy(dev_n, numbers, N * sizeof(T), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	int ThreadsPerBlocks = 1024;
	int BlocksPerGrid = 65536;
	int BlocksAmount = (N%ThreadsPerBlocks == 0) ? N / ThreadsPerBlocks : N / ThreadsPerBlocks + 1;
	// Launch a kernel on the GPU with one thread for each element.
	LeibnitzFormulaGPU << <BlocksAmount, ThreadsPerBlocks >> > (dev_n, N, 0);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "LeibnitzFormulaGPU launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching LeibnitzFormulaGPU!\n", cudaStatus);
		goto Error;
	}
	//}
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(numbers, dev_n, N * sizeof(T), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_n);
	return cudaStatus;
}

template <typename  T>
hipError_t Sum(T* numbers, int N, T SumCPU, LARGE_INTEGER TimeCPU)
{
	T *dev_n = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}


	cudaStatus = hipMalloc((void**)&dev_n, N * sizeof(T));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_n, numbers, N * sizeof(T), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy numbers failed!");
		goto Error;
	}



	int ThreadsPerBlocks = 32;		//<-rozmiar bloku
	int BlocksPerGrid = 65536;	//<-rozmiar siatki
	int SamplesAmount = N;		//<-ilo�� pr�bek
	int BlocksAmount = 0;		//<-liczba blok�w
	int GridsAmount = 0;		//<-liczba siatek


	LARGE_INTEGER StartGPU, StopGPU, TimeGPU;
	LARGE_INTEGER frequency;
	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&StartGPU);
	for (;;) {
		BlocksAmount = (SamplesAmount%ThreadsPerBlocks == 0) ? SamplesAmount / ThreadsPerBlocks : SamplesAmount / ThreadsPerBlocks + 1;	// <-liczba blok�w
		GridsAmount = (BlocksAmount%BlocksPerGrid == 0) ? BlocksAmount / BlocksPerGrid : BlocksAmount / BlocksPerGrid + 1;			// <-Liczenie siatek
		for (int i = 0; i < GridsAmount; i++) {
			SumGPU << <BlocksPerGrid, ThreadsPerBlocks, ThreadsPerBlocks * sizeof(T) >> > (dev_n, SamplesAmount, i*BlocksPerGrid);
		}
		//SamplesAmount = BlocksAmount2;																	//<-Ka�dy blok daje sum�, a te trzeba zsumowa�, wi�c stanowi� one nowe pr�bki
		SamplesAmount = BlocksAmount;
		if (BlocksAmount == 1)																		//<- Je�li liczba blok�w by�a r�wna 1 to znaczy, �e wszystkie pr�bki si� zsumowa�y
			break;
	}
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "SumGPU launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching SumGPU!\n", cudaStatus);
		goto Error;
	}

	QueryPerformanceCounter(&StopGPU);
	TimeGPU.QuadPart = StopGPU.QuadPart - StartGPU.QuadPart;
	TimeGPU.QuadPart = TimeGPU.QuadPart * 1000000 / frequency.QuadPart / 1000;	//ms


	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(numbers, dev_n, N * sizeof(T), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cout << setprecision(50) << "PI=\t\t3.141592653589793238462643383279502884197169399375105820" << endl;
	cout << "SumGPU= \t" << numbers[0] << endl;
	cout << "Czas dodawania na GPU wynosi: " << TimeGPU.QuadPart << "ms" << endl;
	cout << endl << "GPU jest " << (T)(TimeCPU.QuadPart / (T)TimeGPU.QuadPart) << "x szybsze od CPU" << endl;
	cout << "Roznica miedzy suma liczona na CPU i GPU wynosi: " << SumCPU - numbers[0] << endl;

Error:
	hipFree(dev_n);

	return cudaStatus;
}
